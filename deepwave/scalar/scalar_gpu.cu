#include "hip/hip_runtime.h"
#include "scalar_gpu.h"

#include <stddef.h>
#include <stdio.h>
#include <string.h>

#include "scalar.h"

__constant__ TYPE fd1[2 * DIM];
__constant__ TYPE fd2[2 * DIM + 1];

#define gpuErrchk(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort) exit(code);
  }
}

/* Dimension-specific definitions
 *
 * LOOP: Loops over shots and spatial dimensions
 * ENDLOOP: Close the loops in LOOP
 * location_index: Convert array of coordinates into index into flat array
 * */
#if DIM == 1

#define LOOP(num_shots, start_z, end_z, start_y, end_y, start_x, end_x) \
  ptrdiff_t shot = blockIdx.y * blockDim.y + threadIdx.y;               \
  ptrdiff_t z = blockIdx.x * blockDim.x + threadIdx.x + start_z;        \
  ptrdiff_t y = 0;                                                      \
  ptrdiff_t x = 0;                                                      \
  if ((shot < num_shots) && (z < end_z)) {
#define ENDLOOP }

inline __device__ ptrdiff_t
location_index(const ptrdiff_t *__restrict__ const arr, const ptrdiff_t shape_y,
               const ptrdiff_t shape_x, const ptrdiff_t index) {
  const ptrdiff_t z = arr[index];

  return z;
}

#elif DIM == 2

#define LOOP(num_shots, start_z, end_z, start_y, end_y, start_x, end_x) \
  ptrdiff_t shot = blockIdx.z * blockDim.z + threadIdx.z;               \
  ptrdiff_t z = blockIdx.y * blockDim.y + threadIdx.y + start_z;        \
  ptrdiff_t y = blockIdx.x * blockDim.x + threadIdx.x + start_y;        \
  ptrdiff_t x = 0;                                                      \
  if ((shot < num_shots) && (z < end_z) && (y < end_y)) {
#define ENDLOOP }

inline __device__ ptrdiff_t
location_index(const ptrdiff_t *__restrict__ const arr, const ptrdiff_t shape_y,
               const ptrdiff_t shape_x, const ptrdiff_t index) {
  const ptrdiff_t z = arr[index * 2];
  const ptrdiff_t y = arr[index * 2 + 1];

  return z * shape_y + y;
}

#elif DIM == 3

#define LOOP(num_shots, start_z, end_z, start_y, end_y, start_x, end_x) \
  ptrdiff_t threadz = blockIdx.z * blockDim.z + threadIdx.z;            \
  ptrdiff_t shot = threadz / (end_z - start_z + 1);                     \
  ptrdiff_t z = threadz % (end_z - start_z + 1) + start_z;              \
  ptrdiff_t y = blockIdx.y * blockDim.y + threadIdx.y + start_y;        \
  ptrdiff_t x = blockIdx.x * blockDim.x + threadIdx.x + start_x;        \
  if ((shot < num_shots) && (z < end_z) && (y < end_y) && (x < end_x)) {
#define ENDLOOP }

inline __device__ ptrdiff_t
location_index(const ptrdiff_t *__restrict__ const arr, const ptrdiff_t shape_y,
               const ptrdiff_t shape_x, const ptrdiff_t index) {
  const ptrdiff_t z = arr[index * 3];
  const ptrdiff_t y = arr[index * 3 + 1];
  const ptrdiff_t x = arr[index * 3 + 2];

  return z * shape_y * shape_x + y * shape_x + x;
}

#else
#error "Must specify the dimension, e.g. -D DIM=1"
#endif /* DIM */

void setup(const TYPE *__restrict__ const fd1_d,
           const TYPE *__restrict__ const fd2_d) {
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(fd1), fd1_d, 2 * DIM * sizeof(TYPE)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(fd2), fd2_d, (2 * DIM + 1) * sizeof(TYPE)));
}

#if DIM == 1

__global__ void propagate_kernel(
    TYPE *__restrict__ const wfn, TYPE *__restrict__ const phizn,
    const TYPE *__restrict__ const wfc, const TYPE *__restrict__ const wfp,
    const TYPE *__restrict__ const phizc, const TYPE *__restrict__ const sigmaz,
    const TYPE *__restrict__ const model, const ptrdiff_t shape_z,
    const ptrdiff_t numel_shot, const ptrdiff_t size_xy,
    const ptrdiff_t num_shots, const ptrdiff_t pmlz0, const ptrdiff_t pmlz1,
    const TYPE dt) {
  LOOP(num_shots, ZPAD, shape_z - ZPAD, YPAD, 1 - YPAD, XPAD, 1 - XPAD);

  ptrdiff_t i = z;
  ptrdiff_t si = shot * numel_shot + i;

  TYPE lap = laplacian(wfc);

  if ((z >= pmlz0 + 2 * ZPAD) && (z < shape_z - pmlz1 - 2 * ZPAD)) {
    /* Update wavefield */
    wfn[si] = model[i] * lap + 2 * wfc[si] - wfp[si];
  } else {
    /* Inside PML region */

    TYPE wfc_z = z_deriv(wfc);
    TYPE phizc_z = z_deriv(phizc);

    /* Update wavefield */
    wfn[si] = 1 / (1 + dt * sigmaz[z] / 2) *
              (model[i] * (lap + phizc_z) + dt * sigmaz[z] * wfp[si] / 2 +
               (2 * wfc[si] - wfp[si]));

    /* Update phi */
    phizn[si] = phizc[si] - dt * sigmaz[z] * (wfc_z + phizc[si]);
  }

  ENDLOOP;
}

#elif DIM == 2

__global__ void propagate_kernel(
    TYPE *__restrict__ const wfn, TYPE *__restrict__ const phizn,
    TYPE *__restrict__ const phiyn, const TYPE *__restrict__ const wfc,
    const TYPE *__restrict__ const wfp, const TYPE *__restrict__ const phizc,
    const TYPE *__restrict__ const sigmaz, const TYPE *__restrict__ const phiyc,
    const TYPE *__restrict__ const sigmay, const TYPE *__restrict__ const model,
    const ptrdiff_t shape_z, const ptrdiff_t shape_y,
    const ptrdiff_t numel_shot, const ptrdiff_t size_x, const ptrdiff_t size_xy,
    const ptrdiff_t num_shots, const ptrdiff_t pmlz0, const ptrdiff_t pmlz1,
    const ptrdiff_t pmly0, const ptrdiff_t pmly1, const TYPE dt) {
  LOOP(num_shots, ZPAD, shape_z - ZPAD, YPAD, shape_y - YPAD, XPAD, 1 - XPAD);

  ptrdiff_t i = z * size_xy + y;
  ptrdiff_t si = shot * numel_shot + i;

  TYPE lap = laplacian(wfc);

  if ((z >= pmlz0 + 2 * ZPAD) && (z < shape_z - pmlz1 - 2 * ZPAD) &&
      (y >= pmly0 + 2 * YPAD) && (y < shape_y - pmly1 - 2 * YPAD)) {
    /* Update wavefield */
    wfn[si] = model[i] * lap + 2 * wfc[si] - wfp[si];
  } else {
    /* Inside PML region */

    TYPE wfc_z = z_deriv(wfc);
    TYPE phizc_z = z_deriv(phizc);
    TYPE wfc_y = y_deriv(wfc);
    TYPE phiyc_y = y_deriv(phiyc);

    /* Update wavefield */
    wfn[si] =
        1 / (1 + dt * (sigmaz[z] + sigmay[y]) / 2) *
        (model[i] * (lap + phizc_z + phiyc_y) +
         dt * (sigmaz[z] + sigmay[y]) * wfp[si] / 2 + (2 * wfc[si] - wfp[si]) -
         dt * dt * sigmaz[z] * sigmay[y] * wfc[si]);

    /* Update phi */
    phizn[si] = phizc[si] -
                dt * (sigmaz[z] * phizc[si] + (sigmaz[z] - sigmay[y]) * wfc_z);
    phiyn[si] = phiyc[si] -
                dt * (sigmay[y] * phiyc[si] + (sigmay[y] - sigmaz[z]) * wfc_y);
  }

  ENDLOOP;
}

#elif DIM == 3

__global__ void propagate_kernel(
    TYPE *__restrict__ const wfn, TYPE *__restrict__ const phizn,
    TYPE *__restrict__ const phiyn, TYPE *__restrict__ const phixn,
    TYPE *__restrict__ const psin, const TYPE *__restrict__ const wfc,
    const TYPE *__restrict__ const wfp, const TYPE *__restrict__ const phizc,
    const TYPE *__restrict__ const sigmaz, const TYPE *__restrict__ const phiyc,
    const TYPE *__restrict__ const sigmay, const TYPE *__restrict__ const phixc,
    const TYPE *__restrict__ const sigmax, const TYPE *__restrict__ const psic,
    const TYPE *__restrict__ const model, const ptrdiff_t shape_z,
    const ptrdiff_t shape_y, const ptrdiff_t shape_x,
    const ptrdiff_t numel_shot, const ptrdiff_t size_x, const ptrdiff_t size_xy,
    const ptrdiff_t num_shots, const ptrdiff_t pmlz0, const ptrdiff_t pmlz1,
    const ptrdiff_t pmly0, const ptrdiff_t pmly1, const ptrdiff_t pmlx0,
    const ptrdiff_t pmlx1, const TYPE dt) {
  LOOP(num_shots, ZPAD, shape_z - ZPAD, YPAD, shape_y - YPAD, XPAD,
       shape_x - XPAD);

  ptrdiff_t i = z * size_xy + y * size_x + x;
  ptrdiff_t si = shot * numel_shot + i;

  TYPE lap = laplacian(wfc);

  if ((z >= pmlz0 + 2 * ZPAD) && (z < shape_z - pmlz1 - 2 * ZPAD) &&
      (y >= pmly0 + 2 * YPAD) && (y < shape_y - pmly1 - 2 * YPAD) &&
      (x >= pmlx0 + 2 * XPAD) && (x < shape_x - pmlx1 - 2 * XPAD)) {
    /* Update wavefield */
    wfn[si] = model[i] * lap + 2 * wfc[si] - wfp[si];
  } else {
    /* Inside PML region */

    TYPE wfc_z = z_deriv(wfc);
    TYPE phizc_z = z_deriv(phizc);
    TYPE wfc_y = y_deriv(wfc);
    TYPE wfc_x = x_deriv(wfc);
    TYPE phiyc_y = y_deriv(phiyc);
    TYPE phixc_x = x_deriv(phixc);
    TYPE psic_z = z_deriv(psic);
    TYPE psic_y = y_deriv(psic);
    TYPE psic_x = x_deriv(psic);

    /* Update wavefield */
    wfn[si] = 1 / (1 + dt * (sigmaz[z] + sigmay[y] + sigmax[x]) / 2) *
              (model[i] * lap +
               dt * dt *
                   (phizc_z + phiyc_y + phixc_x -
                    sigmaz[z] * sigmay[y] * sigmax[x] * psic[si]) +
               dt * (sigmaz[z] + sigmay[y] + sigmax[x]) * wfp[si] / 2 +
               (2 * wfc[si] - wfp[si]) -
               dt * dt * wfc[si] *
                   (sigmax[x] * sigmay[y] + sigmay[y] * sigmaz[z] +
                    sigmax[x] * sigmaz[z]));

    /* Update phi */
    phizn[si] = phizc[si] - dt * sigmaz[z] * phizc[si] +
                model[i] / dt * (sigmay[y] + sigmax[x]) * wfc_z +
                dt * sigmax[x] * sigmay[y] * psic_z;
    phiyn[si] = phiyc[si] - dt * sigmay[y] * phiyc[si] +
                model[i] / dt * (sigmaz[z] + sigmax[x]) * wfc_y +
                dt * sigmax[x] * sigmaz[z] * psic_y;
    phixn[si] = phixc[si] - dt * sigmax[x] * phixc[si] +
                model[i] / dt * (sigmaz[z] + sigmay[y]) * wfc_x +
                dt * sigmaz[z] * sigmay[y] * psic_x;

    /* Update psi */
    psin[si] = psic[si] + dt * wfc[si];

    ENDLOOP;
  }
}

#endif /* DIM */

void propagate(TYPE *__restrict__ const wfn,        /* next wavefield */
               TYPE *__restrict__ const auxn,       /* next auxiliary */
               const TYPE *__restrict__ const wfc,  /* current wavefield */
               const TYPE *__restrict__ const wfp,  /* previous wavefield */
               const TYPE *__restrict__ const auxc, /* current auxiliary */
               const TYPE *__restrict__ const sigma,
               const TYPE *__restrict__ const model,
               const TYPE *__restrict__ const fd1_d, /* 1st diff coeffs */
               const TYPE *__restrict__ const fd2_d, /* 2nd diff coeffs */
               const ptrdiff_t *__restrict__ const shape,
               const ptrdiff_t *__restrict__ const pml_width,
               const ptrdiff_t num_shots, const TYPE dt) {
  const ptrdiff_t numel_shot = shape[0] * shape[1] * shape[2];
  const ptrdiff_t size_x = shape[2];
  const ptrdiff_t size_xy = shape[1] * shape[2];
  TYPE *__restrict__ const phizn = auxn;
  const TYPE *__restrict__ const phizc = auxc;
  const TYPE *__restrict__ const sigmaz = sigma;

#if DIM >= 2

  TYPE *__restrict__ const phiyn = auxn + num_shots * numel_shot;
  const TYPE *__restrict__ const phiyc = auxc + num_shots * numel_shot;
  const TYPE *__restrict__ const sigmay = sigma + shape[0];

#endif /* DIM >= 2 */

#if DIM == 3

  TYPE *__restrict__ const phixn = auxn + 2 * num_shots * numel_shot;
  TYPE *__restrict__ const psin = auxn + 3 * num_shots * numel_shot;
  const TYPE *__restrict__ const phixc = auxc + 2 * num_shots * numel_shot;
  const TYPE *__restrict__ const psic = auxc + 3 * num_shots * numel_shot;
  const TYPE *__restrict__ const sigmax = sigma + shape[0] + shape[1];

#endif /* DIM  == 3 */

  dim3 dimBlock(32, 32, 1);
#if DIM == 1
  int gridx = (shape[0] - (2 * ZPAD) + dimBlock.x - 1) / dimBlock.x;
  int gridy = (num_shots + dimBlock.y - 1) / dimBlock.y;
  int gridz = 1;
  dim3 dimGrid(gridx, gridy, gridz);
  propagate_kernel<<<dimGrid, dimBlock>>>(
      wfn, phizn, wfc, wfp, phizc, sigmaz, model, shape[0], numel_shot, size_xy,
      num_shots, pml_width[0], pml_width[1], dt);
#elif DIM == 2
  int gridx = (shape[1] - (2 * YPAD) + dimBlock.x - 1) / dimBlock.x;
  int gridy = (shape[0] - (2 * ZPAD) + dimBlock.y - 1) / dimBlock.y;
  int gridz = (num_shots + dimBlock.z - 1) / dimBlock.z;
  dim3 dimGrid(gridx, gridy, gridz);
  propagate_kernel<<<dimGrid, dimBlock>>>(
      wfn, phizn, phiyn, wfc, wfp, phizc, sigmaz, phiyc, sigmay, model,
      shape[0], shape[1], numel_shot, size_x, size_xy, num_shots, pml_width[0],
      pml_width[1], pml_width[2], pml_width[3], dt);
#elif DIM == 3
  int gridx = (shape[2] - (2 * XPAD) + dimBlock.x - 1) / dimBlock.x;
  int gridy = (shape[1] - (2 * YPAD) + dimBlock.y - 1) / dimBlock.y;
  int gridz =
      (num_shots * (shape[0] - (2 * ZPAD)) + dimBlock.z - 1) / dimBlock.z;
  dim3 dimGrid(gridx, gridy, gridz);
  propagate_kernel<<<dimGrid, dimBlock>>>(
      wfn, phizn, phiyn, phixn, psin, wfc, wfp, phizc, sigmaz, phiyc, sigmay,
      phixc, sigmax, psic, model, shape[0], shape[1], shape[2], numel_shot,
      size_x, size_xy, num_shots, pml_width[0], pml_width[1], pml_width[2],
      pml_width[3], pml_width[4], pml_width[5], dt);
#endif /* DIM */

  gpuErrchk(hipPeekAtLastError());
}

void __global__ add_sources_kernel(
    TYPE *__restrict__ const next_wavefield,
    const TYPE *__restrict__ const model,
    const TYPE *__restrict__ const source_amplitudes,
    const ptrdiff_t *__restrict__ const source_locations,
    const ptrdiff_t shape_z, const ptrdiff_t shape_y, const ptrdiff_t shape_x,
    const ptrdiff_t num_shots, const ptrdiff_t num_sources_per_shot) {
  ptrdiff_t shot = blockIdx.y * blockDim.y + threadIdx.y;
  ptrdiff_t source = blockIdx.x * blockDim.x + threadIdx.x;

  if ((shot < num_shots) && (source < num_sources_per_shot)) {
    ptrdiff_t s = shot * num_sources_per_shot + source;
    ptrdiff_t i = location_index(source_locations, shape_y, shape_x, s);
    ptrdiff_t si = shot * shape_z * shape_y * shape_x + i;
    atomicAdd(next_wavefield + si, source_amplitudes[s] * model[i]);
  }
}

void add_sources(TYPE *__restrict__ const next_wavefield,
                 const TYPE *__restrict__ const model,
                 const TYPE *__restrict__ const source_amplitudes,
                 const ptrdiff_t *__restrict__ const source_locations,
                 const ptrdiff_t *__restrict__ const shape,
                 const ptrdiff_t num_shots,
                 const ptrdiff_t num_sources_per_shot) {
  dim3 dimBlock(32, 1, 1);
  int gridx = (num_sources_per_shot + dimBlock.x - 1) / dimBlock.x;
  int gridy = (num_shots + dimBlock.y - 1) / dimBlock.y;
  int gridz = 1;
  dim3 dimGrid(gridx, gridy, gridz);

  add_sources_kernel<<<dimGrid, dimBlock>>>(
      next_wavefield, model, source_amplitudes, source_locations, shape[0],
      shape[1], shape[2], num_shots, num_sources_per_shot);

  gpuErrchk(hipPeekAtLastError());
}

void __global__ record_receivers_kernel(
    TYPE *__restrict__ const receiver_amplitudes,
    const TYPE *__restrict__ const current_wavefield,
    const ptrdiff_t *__restrict__ const receiver_locations,
    const ptrdiff_t shape_z, const ptrdiff_t shape_y, const ptrdiff_t shape_x,
    const ptrdiff_t num_shots, const ptrdiff_t num_receivers_per_shot) {
  ptrdiff_t shot = blockIdx.y * blockDim.y + threadIdx.y;
  ptrdiff_t receiver = blockIdx.x * blockDim.x + threadIdx.x;

  if ((shot < num_shots) && (receiver < num_receivers_per_shot)) {
    ptrdiff_t r = shot * num_receivers_per_shot + receiver;
    ptrdiff_t si = shot * shape_z * shape_y * shape_x +
                   location_index(receiver_locations, shape_y, shape_x, r);
    receiver_amplitudes[r] = current_wavefield[si];
  }
}

void record_receivers(TYPE *__restrict__ const receiver_amplitudes,
                      const TYPE *__restrict__ const current_wavefield,
                      const ptrdiff_t *__restrict__ const receiver_locations,
                      const ptrdiff_t *__restrict__ const shape,
                      const ptrdiff_t num_shots,
                      const ptrdiff_t num_receivers_per_shot) {
  if (receiver_amplitudes == NULL) return; /* no source inversion */

  dim3 dimBlock(32, 1, 1);
  int gridx = (num_receivers_per_shot + dimBlock.x - 1) / dimBlock.x;
  int gridy = (num_shots + dimBlock.y - 1) / dimBlock.y;
  int gridz = 1;
  dim3 dimGrid(gridx, gridy, gridz);

  record_receivers_kernel<<<dimGrid, dimBlock>>>(
      receiver_amplitudes, current_wavefield, receiver_locations, shape[0],
      shape[1], shape[2], num_shots, num_receivers_per_shot);

  gpuErrchk(hipPeekAtLastError());
}

void save_wavefields(TYPE *__restrict__ const saved_wavefields,
                     const TYPE *__restrict__ const current_wavefield,
                     const ptrdiff_t *__restrict__ const shape,
                     const ptrdiff_t num_shots, const ptrdiff_t step,
                     const enum wavefield_save_strategy save_strategy) {
  if (save_strategy == STRATEGY_COPY) {
    TYPE *__restrict__ current_saved_wavefield = set_step_pointer(
        saved_wavefields, step, num_shots, shape[0] * shape[1] * shape[2]);
    gpuErrchk(
        hipMemcpy(current_saved_wavefield, current_wavefield,
                   num_shots * shape[0] * shape[1] * shape[2] * sizeof(TYPE),
                   hipMemcpyDeviceToDevice));
  }
}

void __global__ imaging_condition_kernel(
    TYPE *__restrict__ const model_grad,
    const TYPE *__restrict__ const current_wavefield,
    const TYPE *__restrict__ const next_adjoint_wavefield,
    const TYPE *__restrict__ const current_adjoint_wavefield,
    const TYPE *__restrict__ const previous_adjoint_wavefield,
    const TYPE *__restrict__ const sigmaz,
    const TYPE *__restrict__ const sigmay,
    const TYPE *__restrict__ const sigmax, const ptrdiff_t shape_z,
    const ptrdiff_t shape_y, const ptrdiff_t shape_x,
    const ptrdiff_t numel_shot, const ptrdiff_t size_x, const ptrdiff_t size_xy,
    const ptrdiff_t num_shots, const TYPE dt) {
  LOOP(num_shots, ZPAD, shape_z - ZPAD, YPAD, shape_y - YPAD, XPAD,
       shape_x - XPAD);

  ptrdiff_t i = z * size_xy + y * size_x + x;
  ptrdiff_t si = shot * numel_shot + i;

  TYPE adjoint_wavefield_tt =
      (next_adjoint_wavefield[si] - 2 * current_adjoint_wavefield[si] +
       previous_adjoint_wavefield[si]) /
      (dt * dt);

  TYPE adjoint_wavefield_t =
      (next_adjoint_wavefield[si] - previous_adjoint_wavefield[si]) / (2 * dt);

#if DIM == 1

  atomicAdd(model_grad + i,
            current_wavefield[si] *
                (adjoint_wavefield_tt + sigmaz[z] * adjoint_wavefield_t));

#elif DIM == 2

  atomicAdd(model_grad + i,
            current_wavefield[si] *
                (adjoint_wavefield_tt +
                 (sigmaz[z] + sigmay[y]) * adjoint_wavefield_t +
                 sigmaz[z] * sigmay[y] * current_adjoint_wavefield[si]));

#elif DIM == 3

  /* NOTE: There should be an additional term here (involving spatial
   * derivative of phi, sigma, and psi), but it is neglected due to
   * the additional computational cost it would cause. */
  atomicAdd(model_grad + i,
            current_wavefield[si] *
                (adjoint_wavefield_tt +
                 (sigmaz[z] + sigmay[y] + sigmax[x]) * adjoint_wavefield_t +
                 (sigmax[x] * sigmay[y] + sigmay[y] * sigmaz[z] +
                  sigmax[x] * sigmaz[z]) *
                     current_adjoint_wavefield[si]));

#endif

  ENDLOOP;
}

void imaging_condition(
    TYPE *__restrict__ const model_grad,
    const TYPE *__restrict__ const current_wavefield,
    const TYPE *__restrict__ const next_adjoint_wavefield,
    const TYPE *__restrict__ const current_adjoint_wavefield,
    const TYPE *__restrict__ const previous_adjoint_wavefield,
    const TYPE *__restrict__ const sigma,
    const ptrdiff_t *__restrict__ const shape,
    const ptrdiff_t *__restrict__ const pml_width, const ptrdiff_t num_shots,
    const TYPE dt) {
  if (model_grad == NULL) return; /* Not doing model inversion */

  dim3 dimBlock(32, 32, 1);
#if DIM == 1
  int gridx = (shape[0] - (2 * ZPAD) + dimBlock.x - 1) / dimBlock.x;
  int gridy = (num_shots + dimBlock.y - 1) / dimBlock.y;
  int gridz = 1;
#elif DIM == 2
  int gridx = (shape[1] - (2 * YPAD) + dimBlock.x - 1) / dimBlock.x;
  int gridy = (shape[0] - (2 * ZPAD) + dimBlock.y - 1) / dimBlock.y;
  int gridz = (num_shots + dimBlock.z - 1) / dimBlock.z;
#elif DIM == 3
  int gridx = (shape[2] - (2 * XPAD) + dimBlock.x - 1) / dimBlock.x;
  int gridy = (shape[1] - (2 * YPAD) + dimBlock.y - 1) / dimBlock.y;
  int gridz =
      (num_shots * (shape[0] - (2 * ZPAD)) + dimBlock.z - 1) / dimBlock.z;
#endif /* DIM */

  dim3 dimGrid(gridx, gridy, gridz);
  const TYPE *__restrict__ const sigmaz = sigma;
  const TYPE *__restrict__ const sigmay = sigma + shape[0];
  const TYPE *__restrict__ const sigmax = sigma + shape[0] + shape[1];

  imaging_condition_kernel<<<dimGrid, dimBlock>>>(
      model_grad, current_wavefield, next_adjoint_wavefield,
      current_adjoint_wavefield, previous_adjoint_wavefield, sigmaz, sigmay,
      sigmax, shape[0], shape[1], shape[2], shape[0] * shape[1] * shape[2],
      shape[2], shape[1] * shape[2], num_shots, dt);

  gpuErrchk(hipPeekAtLastError());
}

void __global__ model_grad_scaling_kernel(
    TYPE *__restrict__ const model_grad, const TYPE *__restrict__ const scaling,
    const ptrdiff_t numel_shot) {
  ptrdiff_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < numel_shot) {
    model_grad[i] *= scaling[i];
  }
}

void model_grad_scaling(TYPE *__restrict__ const model_grad,
                        const TYPE *__restrict__ const scaling,
                        const ptrdiff_t *__restrict__ const shape,
                        const ptrdiff_t *__restrict__ const pml_width) {
  if (model_grad == NULL) return; /* Not doing model inversion */

  const ptrdiff_t numel_shot = shape[0] * shape[1] * shape[2];

  dim3 dimBlock(32, 1, 1);
  int gridx = (numel_shot + dimBlock.x - 1) / dimBlock.x;
  int gridy = 1;
  int gridz = 1;
  dim3 dimGrid(gridx, gridy, gridz);

  model_grad_scaling_kernel<<<dimGrid, dimBlock>>>(model_grad, scaling,
                                                   numel_shot);

  gpuErrchk(hipPeekAtLastError());
}
